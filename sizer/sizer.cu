#include "hip/hip_runtime.h"
#include <stdio.h>
#include <boost/multi_array.hpp>
#include <H5Cpp.h>

#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#include "hip/hip_math_constants.h"

using boost::multi_array;
using boost::extents;

const int NX = 414;
const int NY = 414;

struct idealsphere : public thrust::unary_function<int, float>
{
  float rfactor;
  float roffset;
  float baseoffsetx;
  float baseoffsety;
  float lfactor;
  float offsetx, offsety;
  float r;
  thrust::device_ptr<short> dPhotons;
  thrust::device_ptr<float> dLambdas;

  idealsphere(thrust::device_vector<short>& dPhotons, thrust::device_vector<float>& dLambdas) :
  dPhotons(dPhotons.data()), dLambdas(dLambdas.data()) {}
  
  __host__ __device__ float operator () (const int& data) {
    if (!dLambdas[data]) return 0;

    float x = (data % NX) - offsetx;
    float y = (data / NX) - offsety;
    
    float q = sqrt(x * x + y * y);
    float val = 3 * (sinpif(2 * q * r)  - 2 * ((float) HIP_PI_F) * q * r * cospif(2 * q * r));
    float den = (2 * ((float) HIP_PI_F) * q * r);
    den = den * den * den;
    
    val /= den;
    val = val * val;
    
    return val;
  }
};

struct likelihood : public thrust::unary_function<int, float>
{
  idealsphere& spherer;
  float factor;
__host__ __device__ likelihood(idealsphere& spherer, float factor) : spherer(spherer), factor(factor) {}

  __host__ __device__ float operator () (const int& data) {
    if (!spherer.dLambdas[data]) return 0;

    float intensity = spherer(data) * factor + spherer.dLambdas[data] * spherer.lfactor;
    float val = 0;
    if (spherer.dPhotons[data])
      {
	val += spherer.dPhotons[data] * log(intensity);
      }
    val -= intensity;

    return val;
  }
};


__global__ void computeintensity(float* target, float* intens, idealsphere myspherer, float psum, float lsum)
{
  myspherer.r = exp(myspherer.roffset + (threadIdx.z + blockIdx.z * blockDim.z) * myspherer.rfactor);
  myspherer.offsetx = (threadIdx.x) * 3 + myspherer.baseoffsetx;
  myspherer.offsety = (threadIdx.y * 3 + myspherer.baseoffsety);
  myspherer.lfactor = 1.0 / sqrt(lsum) * (((int) blockIdx.x)) + 1.0;
//    myspherer.lfactor = 1.0;

  int idx =  (threadIdx.x + blockIdx.x * blockDim.x)  + (threadIdx.y + blockIdx.y * blockDim.y) * gridDim.x * blockDim.x + (threadIdx.z + blockIdx.z * blockDim.z)  * gridDim.x * blockDim.x * gridDim.y * blockDim.y;
  float intensity = thrust::reduce(thrust::seq,
				   thrust::make_transform_iterator(thrust::make_counting_iterator(0), myspherer),
				   thrust::make_transform_iterator(thrust::make_counting_iterator(NY * NX), myspherer));

  float intensityfactor = (psum - lsum * myspherer.lfactor) / intensity;
  if (intensityfactor < 1e-6)
{
/*target[idx] = -1e10;
intens[idx] = 0;*/
intensityfactor = 1e-6;
}
  intensityfactor *= pow(1.01, blockIdx.y - gridDim.y * 0.5);
  likelihood likelihooder(myspherer, intensityfactor);
  float likelihood1 = thrust::reduce(thrust::seq,
				   thrust::make_transform_iterator(thrust::make_counting_iterator(0), likelihooder),
				   thrust::make_transform_iterator(thrust::make_counting_iterator(NY * NX), likelihooder));

  target[idx] = likelihood1;
  intens[idx] = likelihooder.factor;
}

int main()
{
//    H5::H5File file("/scratch/fhgfs/alberto/MPI/TODO/EXPERIMENTAL/MASK_90000px/May2013_RNApol/Background_RNA/HITS_RNApol/ToFhits.h5", H5F_ACC_RDONLY);
//  H5::H5File file("/scratch/fhgfs/alberto/MPI/TODO/EXPERIMENTAL/MASK_90000px/May2013_MS2/ALL_runsRNApol_differentGainmapAndMask/HITS354/HITS.h5", H5F_ACC_RDONLY);
//H5::H5File file("/scratch/fhgfs/alberto/MPI/TODO/EXPERIMENTAL/MASK_90000px/May2013_MS2/ALL_runsMS2/HITS_MS2/HITS.h5", H5F_ACC_RDONLY);
//H5::H5File file("/scratch/fhgfs/alberto/MPI/TODO/EXPERIMENTAL/MASK_90000px/May2013_MS2/ALL_runsTBSV/HITS_TBSV/HITS.h5", H5F_ACC_RDONLY);
H5::H5File file("/scratch/fhgfs/alberto/MPI/TODO/EXPERIMENTAL/MASK_90000px/May2013_RNApol/BackgroundProva/HITS_RNApol/Hits.h5", H5F_ACC_RDONLY);
//  H5::H5File file("/scratch/fhgfs/alberto/MPI/TODO/EXPERIMENTAL/MASK_90000px/May2013_RNApol/12nm_SIM/HITS/HITS.h5", H5F_ACC_RDONLY);
//  H5::H5File file("/scratch/fhgfs/alberto/MPI/TODO/EXPERIMENTAL/MASK_90000px/20sizes_RNA/forCARL_PDB/HITS_PDB.h5", H5F_ACC_RDONLY);
  H5::Group group = file.openGroup("with_geometry");
  H5::DataSet lambdas = group.openDataSet("lambdas");
  H5::DataSet photons = group.openDataSet("photon_count");
  H5::DataSpace lambdaSpace = lambdas.getSpace();
  H5::DataSpace photonSpace = photons.getSpace();
  
  hsize_t count[3] = {1, NY, NX};
  hsize_t offset[3] = {0, 0, 0};
  lambdaSpace.selectHyperslab(H5S_SELECT_SET, count, offset);
  photonSpace.selectHyperslab(H5S_SELECT_SET, count, offset);
  H5::DataSpace memSpace(3, count);

  boost::multi_array<float, 2> lambdaVals(extents[NY][NX]);
  boost::multi_array<short, 2> photonVals(extents[NY][NX]);
  thrust::device_vector<short> dPhotons(NY * NX);
  thrust::device_vector<float> dLambdas(NY * NX);
  thrust::device_vector<float> dIntensity(175000000);
  thrust::host_vector<float> hIntensity(175000000);
  thrust::device_vector<float> dIntensity2(175000000);
  thrust::host_vector<float> hIntensity2(175000000);

  idealsphere spherer(dPhotons, dLambdas);

  for (int img = 0; img < 2779; img++)
    {
      offset[0] = img;
      lambdaSpace.selectHyperslab(H5S_SELECT_SET, count, offset);
      photonSpace.selectHyperslab(H5S_SELECT_SET, count, offset);
      lambdas.read(lambdaVals.data(), H5::PredType::NATIVE_FLOAT, memSpace, lambdaSpace);
      photons.read(photonVals.data(), H5::PredType::NATIVE_SHORT, memSpace, photonSpace);

      int psum = 0;
      double lsum = 0;
      for (int y = 0; y < NY; y++)
	{
	  for (int x = 0; x < NX; x++)
	    {
	      if (((y > 195 && y < 231) || y < 36 || ((x < 255  || (x < 300 && (y > 124 && y < 325))) && (y > 92 || x > 170 || x < 90)) && !(x<55 && y > 374))
	      || y < 105 || y > 343)
	      {
		photonVals[y][x] = 0;
		lambdaVals[y][x] = 0;
	      }
	      psum += photonVals[y][x];
	      lsum += lambdaVals[y][x];
	    }
	}
      
      dim3 grid(1, 50, 1200);
      dim3 block(32, 32, 1);

      spherer.rfactor = 0.005;
      spherer.roffset = -10;
      spherer.baseoffsetx = NX / 2 - 10 - 51 - 0.5; // good val -10
      spherer.baseoffsety = NY / 2 + 10 - 51 - 0.5; // good val +10
      dPhotons.assign(photonVals.data(), photonVals.data() + NY * NX);
      dLambdas.assign(lambdaVals.data(), lambdaVals.data() + NY * NX);
      
      computeintensity<<<grid, block>>>(dIntensity.data().get(), dIntensity2.data().get(), spherer, psum, lsum);
      hIntensity.assign(dIntensity.begin(), dIntensity.end());
      hIntensity2.assign(dIntensity2.begin(), dIntensity2.end());

      float minval = 1e30;
float maxval = -1e30;
      int maxidx = 0;
      float maxint = 0;
      for (int k = 0; k < grid.y * block.y * grid.x * block.x * grid.z * block.z; k++)
{
	if (hIntensity[k] > maxval)
	{
		maxval = hIntensity[k];
		maxint = hIntensity2[k];
		maxidx = k;
	}
	if (hIntensity[k] < minval) minval = hIntensity[k];
}
	int maxR = maxidx / grid.y / block.y / grid.x / block.x;
	int maxX = maxidx % block.x;
	int maxY = (maxidx / (grid.x * block.x)) % block.y;
	int maxI = (maxidx / (block.x * grid.x * block.y)) % grid.y;
	int maxI2 = (maxidx / block.x) % grid.x;

      printf("%d %d %lf %g %g %g %d %d %d %d %g %d %d\n", img, psum, lsum, minval, maxval, hIntensity[0], maxR, maxX, maxY, hipGetLastError(), maxint, maxI, maxI2);
      fflush(stdout);
    }
}
